#include "hip/hip_runtime.h"
//
// Created by yuliana on 24.02.25.
//

#include "riemann_sum_cuda.h"
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
__device__ double func(double x, double y) {
    double sums = 0.0;

    for (int i = -2; i <= 2; i++) {
        for (int j = -2; j <= 2; j++){
            sums += 1.0 / (5 * (i+2) + j + 3 + pow( x - 16*j , 6) + pow( y - 16*i , 6));
        }
    }
    return 1.0 / (0.002 + sums);
}

__global__ void riemannSum(double x1, double x2,
                             double y1, double y2,
                             int steps,
                             double *result) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    double dx = (x2 - x1) / steps;
    double dy = (y2 - y1) / steps;

    if (i < steps && j < steps) {
        double x = x1 + i * dx;
        double y = y1 + j * dy;
        double f_val = func(x, y) * dx * dy;

        atomicAdd(result, f_val);

        //        printf("Thread (%d, %d) computing x = %f, y = %f, localSum = %f\n", i, j, x, y, f_val);
    }
}

double riemman_return(double x1, double x2, double y1, double y2, int steps){
    double h_result = 0.0, *d_result;
    hipMalloc((void**)&d_result, sizeof(double));
    hipMemcpy(d_result, &h_result, sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((steps + BLOCK_SIZE - 1) / BLOCK_SIZE, (steps + BLOCK_SIZE - 1) / BLOCK_SIZE);

    riemannSum<<<gridSize, blockSize>>>(x1, x2, y1, y2, steps, d_result);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_result);

    return h_result;
}

// int main(){
//     int n = 1000;
//     double result = riemman_return(-50, 50, -50, 50, n);
//     std::cout << "Result: " << result << std::endl;
// }