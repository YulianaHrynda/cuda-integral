#include "hip/hip_runtime.h"
//
// Created by yuliana on 24.02.25.
//

#include "riemann_sum_cuda.h"
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>

#include "functions.h"

#define BLOCK_SIZE 16
// __device__ double func(double x, double y) {
//     double sums = 0.0;
//
//     for (int i = -2; i <= 2; i++) {
//         for (int j = -2; j <= 2; j++){
//             sums += 1.0 / (5 * (i+2) + j + 3 + pow( x - 16*j , 6) + pow( y - 16*i , 6));
//         }
//     }
//     return 1.0 / (0.002 + sums);
// }

__device__ double call_function(int func_id, double x, double y) {
    switch (func_id) {
        case 1: return func_1(x, y);
        case 2: return func_2(x, y);
        case 3: return func_3(x, y);
    default: return 0.0;
    }
}


__global__ void riemannSum(double x1, double x2,
                           double y1, double y2,
                           int steps,
                           double *result,
                           int func_id) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    double dx = (x2 - x1) / steps;
    double dy = (y2 - y1) / steps;

    if (i < steps && j < steps) {
        double x = x1 + i * dx;
        double y = y1 + j * dy;
        double f_val = call_function(func_id, x, y) * dx * dy;

        atomicAdd(result, f_val);
    }
}


double riemman_return(double x1, double x2, double y1, double y2, int steps, int func_id) {
    double h_result = 0.0, *d_result;
    hipMalloc((void**)&d_result, sizeof(double));
    hipMemcpy(d_result, &h_result, sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((steps + 15) / 16, (steps + 15) / 16);

    riemannSum<<<gridSize, blockSize>>>(x1, x2, y1, y2, steps, d_result, func_id);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_result);

    return h_result;
}

