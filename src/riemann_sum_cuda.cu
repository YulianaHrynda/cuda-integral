#include "hip/hip_runtime.h"
#include "riemann_sum_cuda.h"
#include <hip/hip_runtime.h>

#include "functions.h"

#define BLOCK_SIZE 16

__device__ double call_function(int func_id, double x, double y) {
    switch (func_id) {
        case 1: return func_1(x, y);
        case 2: return func_2(x, y);
        case 3: return func_3(x, y);
    default: return 0.0;
    }
}


__global__ void riemannSum(double x1, double x2,
                           double y1, double y2,
                           int steps,
                           double *result,
                           int func_id) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    double dx = (x2 - x1) / steps;
    double dy = (y2 - y1) / steps;

    if (i < steps && j < steps) {
        double x = x1 + i * dx;
        double y = y1 + j * dy;
        double f_val = call_function(func_id, x, y) * dx * dy;

        atomicAdd(result, f_val);
    }
}


double riemman_return(double x1, double x2, double y1, double y2, int steps, int func_id) {
    double h_result = 0.0, *d_result;
    hipMalloc((void**)&d_result, sizeof(double));
    hipMemcpy(d_result, &h_result, sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((steps + 15) / 16, (steps + 15) / 16);

    riemannSum<<<gridSize, blockSize>>>(x1, x2, y1, y2, steps, d_result, func_id);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_result);

    return h_result;
}

