#include "hip/hip_runtime.h"
//
// Created by gllekk on 25.02.25.
//

#include "functions.h"

__device__ double func_1(const double x, const double y) {
    double sums = 0.0;
    for (int i = -2; i <= 2; i++) {
        for (int j = -2; j <= 2; j++) {
            sums += 1.0 / (5 * (i + 2) + j + 3 + std::pow(x - 16 * j, 6) + std::pow(y - 16 * i, 6));
        }
    }
    return 1.0 / (0.002 + sums);
}

__device__ double func_2(const double x, const double y) {
    const double a = 20.0;
    const double b = 0.2;
    const double c = 2 * M_PI;

    return -a * std::exp(-b * std::sqrt(0.5 * (x * x + y * y)))
           - std::exp(0.5 * (std::cos(c * x) + std::cos(c * y)))
           + a + std::exp(1.0);
}

__device__ double func_3(const double x, const double y) {
    const int m = 5;
    const double a1[m] = {1, 2, 1, 1, 5};
    const double a2[m] = {4, 5, 1, 2, 4};
    const double c[m]  = {2, 1, 4, 7, 2};

    double result = 0.0;

    for (int i = 0; i < m; ++i) {
        double val = (x - a1[i]) * (x - a1[i]) + (y - a2[i]) * (y - a2[i]);
        result += c[i] * exp(-val / M_PI) * cos(M_PI * val);
    }

    return -result;
}

